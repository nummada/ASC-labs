#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils/utils.h"

// ~TODO 3~
// Modify the kernel below such as each element of the 
// array will be now equal to 0 if it is an even number
// or 1, if it is an odd number
__global__ void kernel_parity_id(int *a, int N) {
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < N) {
		if(a[i] % 2 == 0) {
			a[i] = 0;
		} else {
			a[i] = 1;
		}
	}
}

// ~TODO 4~
// Modify the kernel below such as each element will
// be equal to the BLOCK ID this computation takes
// place.
__global__ void kernel_block_id(int *a, int N) {
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < N) {
		a[i] = blockIdx.x;
	}
}

// ~TODO 5~
// Modify the kernel below such as each element will
// be equal to the THREAD ID this computation takes
// place.
__global__ void kernel_thread_id(int *a, int N) {
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < N) {
		a[i] = threadIdx.x ;
	}
}

int main(void) {
    int nDevices;

    // Get the number of CUDA-capable GPU(s)
    hipGetDeviceCount(&nDevices);

	// ~TODO 1~
	// For each device, show some details in the format below, 
	// then set as active device the first one (assuming there
	// is at least CUDA-capable device). Pay attention to the
	// type of the fields in the hipDeviceProp_t structure.
	//
	// Device number: <i>
	//      Device name: <name>
	//      Total memory: <mem>
	//      Memory Clock Rate (KHz): <mcr>
	//      Memory Bus Width (bits): <mbw>
	// 
	// Hint: look for hipGetDeviceProperties and hipSetDevice in
	// the Cuda Toolkit Documentation.
	hipDeviceProp_t prop;
	for (int i = 0; i < nDevices; ++i) {
		hipGetDeviceProperties( &prop, i);
		printf("Device number: <%d>\n", i);
		printf("\tDevice name: <%s>\n", prop.name);
		printf("\tTotal memory: <%u>\n", prop.totalGlobalMem );
		printf("\tMemory clock rate (KHz): %d\n", prop.clockRate);
		printf("\tMemory bus width: (bits) %d\n", prop.memoryBusWidth);
	}
	hipSetDevice(0);

	// ~TODO 2~
	// With information from example_2.cu, allocate an array with
	// integers (where a[i] = i). Then, modify the three kernels
	// above and execute them using 4 blocks, each with 4 threads.
	// Hint: num_elements = block_size * block_no (see example_2)
	//
	// You can use the fill_array_int(int *a, int n) function (from utils)
	// to fill your array as many times you want.

    
	const int num_elements = 16;
	const int num_bytes = num_elements * sizeof(int);

	int *host_array_a = 0;
	int *host_array_b = 0;

	int *device_array = 0;


	host_array_a = (int *) malloc(num_bytes);
	host_array_b = (int *) malloc(num_bytes);

	hipMalloc((void **) &device_array, num_bytes);


	for (int i = 0; i < num_elements; ++i) {
		host_array_a[i] = (int) i;
	}

	
	
	const size_t block_size = 4;
	size_t blocks_no = num_elements / block_size;
	
	if (num_elements % block_size) 
	++blocks_no;
	
	// ~TODO 3~
	// Execute kernel_parity_id kernel and then copy from 
	// the device to the host; call hipDeviceSynchronize()
	// after a kernel execution for safety purposes.
	//
	hipMemcpy(device_array, host_array_a, num_bytes, hipMemcpyHostToDevice);
    kernel_parity_id<<<blocks_no, block_size>>>(device_array, num_elements);

	hipDeviceSynchronize();
	hipMemcpy(host_array_b, device_array, num_bytes, hipMemcpyDeviceToHost);
	check_task_1(3, host_array_b);

	// ~TODO 4~
	// Execute kernel_block_id kernel and then copy from 
	// the device to the host;
	//
	hipMemcpy(device_array, host_array_a, num_bytes, hipMemcpyHostToDevice);
	kernel_block_id<<<blocks_no, block_size>>>(device_array, num_elements);
	hipDeviceSynchronize();
	hipMemcpy(host_array_b, device_array, num_bytes, hipMemcpyDeviceToHost);

	check_task_1(4, host_array_b);

	// ~TODO 5~
	// Execute kernel_thread_id kernel and then copy from 
	// the device to the host;
	//

	hipMemcpy(device_array, host_array_a, num_bytes, hipMemcpyHostToDevice);
	kernel_thread_id<<<blocks_no, block_size>>>(device_array, num_elements);
	hipDeviceSynchronize();
	hipMemcpy(host_array_b, device_array, num_bytes, hipMemcpyDeviceToHost);

	check_task_1(5, host_array_b);

    // TODO 6: Free the memory
	free(host_array_a);
	free(host_array_b);
	hipFree(device_array);
	return 0;
}